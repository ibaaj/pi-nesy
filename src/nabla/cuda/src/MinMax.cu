#include <stdio.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__global__ void MinMaxKernel(
    float *A,    // Matrix A
    float *x,    // Vector x
    float *result, // Output result, one element per row of A
    int n,       // Number of rows in A
    int m,       // Number of columns in A
    float epsilon // Small threshold for numerical stability, not used here
) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n) {
        float minMax = FLT_MAX; 
        for (int col = 0; col < m; ++col) {
            float currentElement = A[row * m + col];
            float vectorElement = x[col];
            float maxVal = fmax(currentElement, vectorElement);
            minMax = fmin(minMax, maxVal);
        }
        result[row] = minMax; 
    }
}

void MinMax(const float* A, const float* x, float* result, int n, int m, float epsilon) {
    float* A_gpu = nullptr;
    float* x_gpu = nullptr;
    float* result_gpu = nullptr;

    hipMalloc(&A_gpu, n * m * sizeof(float));
    hipMalloc(&x_gpu, m * sizeof(float));
    hipMalloc(&result_gpu, n * sizeof(float));

    hipMemcpy(A_gpu, A, n * m * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(x_gpu, x, m * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(512); // Adjust this based on the device capabilities
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    
    hipEventRecord(start);
    MinMaxKernel<<<gridSize, blockSize>>>(A_gpu, x_gpu, result_gpu, n, m, epsilon);
    
    hipEventRecord(stop);

    
    hipEventSynchronize(stop);

   
    hipMemcpy(result, result_gpu, n * sizeof(float), hipMemcpyDeviceToHost);

    
    hipFree(A_gpu);
    hipFree(x_gpu);
    hipFree(result_gpu);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
